//need to add in other stuff like deinitializing memory and also like the cuda runtime stuff

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matMul(float *d_A, float *d_B, float *d_C, int M, int N, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < P) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += d_A[row * N + i] * d_B[i * P + col];
        }
        d_C[row * P + col] = sum;
    }
}

int main() {
    int M = 16, N = 16, P = 16; // Modified dimensions to match your tiled example
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    h_A = (float*)malloc(M * N * sizeof(float));
    h_B = (float*)malloc(N * P * sizeof(float));
    h_C = (float*)malloc(M * P * sizeof(float));

    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_B, N * P * sizeof(float));
    hipMalloc((void**)&d_C, M * P * sizeof(float));

    // Initialization (Same as your original code)
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            h_A[i * N + j] = (float)(rand() % 10 + 1);
        }
    }
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
            h_B[i * P + j] = (float)(rand() % 10 + 1);
        }
    }

    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * P * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((P + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    matMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, P);
    hipMemcpy(h_C, d_C, M * P * sizeof(float), hipMemcpyDeviceToHost);

    // Print and cleanup code (Same as your original code)
    printf("--------\n");
    printf("Matrix A:\n--------\n");
    for(int i = 0; i < M; ++i) {
            for(int j = 0; j < N; ++j) {
                    printf("%f ", h_A[i * N + j]);
            }
            printf("\n");
    }
      
    printf("--------\n");
    printf("Matrix B:\n--------\n");
    for(int i = 0; i < N; ++i) {
            for(int j = 0; j < P; ++j) {
                    printf("%f ", h_B[i * P + j]);
            }
            printf("\n");
    }
        
    printf("--------\n");
    printf("Matrix C:\n--------\n");
    for(int i = 0; i < M; ++i) {
            for(int j = 0; j < P; ++j) {
                    printf("%f ", h_C[i * P + j]);
            }
            printf("\n");
    }
    printf("--------\n");
}
