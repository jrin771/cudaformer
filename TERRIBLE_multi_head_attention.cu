//code here sucks will rewrite once fundamentals are all done and explained well  



#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

const int num_heads = 2;

__global__ void fused_multi_head_attention(
    float* q, float* k, float* v,
    float* wq, float* wk, float* wv, float* wo,
    float* output, int T, int D
) {
    extern __shared__ float smem[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int t = tid / D;
    int d = tid % D;

    int head_dim = D / num_heads;

    float* dot_products = smem;
    float* softmax_weights = smem + T;

    if (tid < T * D) {
        float multi_head_sum = 0.0f;
        for (int h = 0; h < num_heads; ++h) {
            float sum = 0.0f;

            if (threadIdx.x < T) {
                float dot_product = 0.0f;
                for (int j = 0; j < head_dim; ++j) {
                    int idx = h * head_dim + j;
                    dot_product += q[t * D + j] * wq[idx] * k[threadIdx.x * D + j] * wk[idx];
                }
                dot_products[threadIdx.x] = dot_product / sqrtf((float)head_dim);
            }
            __syncthreads();

            if (threadIdx.x == 0) {
                float max_val = dot_products[0];
                for (int i = 1; i < T; ++i) {
                    max_val = fmaxf(max_val, dot_products[i]);
                }
                float exp_sum = 0.0f;
                for (int i = 0; i < T; ++i) {
                    softmax_weights[i] = expf(dot_products[i] - max_val);
                    exp_sum += softmax_weights[i];
                }
                for (int i = 0; i < T; ++i) {
                    softmax_weights[i] /= exp_sum;
                }
            }
            __syncthreads();

            if (threadIdx.x < T) {
                sum = softmax_weights[threadIdx.x] * v[threadIdx.x * D + d] * wv[h * head_dim + d];
            }
            __syncthreads();

            atomicAdd(&multi_head_sum, sum * wo[d]);
        }
        
        output[tid] = multi_head_sum;
    }
}


//I might just get rid of this since this is probably overkill. 

// LCG values for a simple PRNG
unsigned long a = 1664525;
unsigned long c = 1013904223;
unsigned long seed = 1;  // Can be any value

float pseudo_rand() {
    seed = (a * seed + c);  // Update the seed for next iteration
    return (seed & 0xFFFFFFF) / (float)0xFFFFFFF;  // Return a float between 0 and 1
}

int main() {
    int T = 4, D = 8;
    int num_elements = T * D;

    // Allocate host memory
    float h_q[num_elements], h_k[num_elements], h_v[num_elements], h_wq[D], h_wk[D], h_wv[D], h_wo[D], h_output[num_elements];

   // Initialize host data with random floats between 0 and 1
    for (int i = 0; i < num_elements; ++i) {
        h_q[i] = pseudo_rand();
        h_k[i] = pseudo_rand();
        h_v[i] = pseudo_rand();
    }
    for (int i = 0; i < D; ++i) {
        h_wq[i] = pseudo_rand();
        h_wk[i] = pseudo_rand();
        h_wv[i] = pseudo_rand();
        h_wo[i] = pseudo_rand();
    }

    // Allocate device memory
    float *d_q, *d_k, *d_v, *d_wq, *d_wk, *d_wv, *d_wo, *d_output;
    hipMalloc((void**)&d_q, num_elements * sizeof(float));
    hipMalloc((void**)&d_k, num_elements * sizeof(float));
    hipMalloc((void**)&d_v, num_elements * sizeof(float));
    hipMalloc((void**)&d_wq, D * sizeof(float));
    hipMalloc((void**)&d_wk, D * sizeof(float));
    hipMalloc((void**)&d_wv, D * sizeof(float));
    hipMalloc((void**)&d_wo, D * sizeof(float));
    hipMalloc((void**)&d_output, num_elements * sizeof(float));

    // Copy host to device
    hipMemcpy(d_q, h_q, num_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k, num_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, num_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wq, h_wq, D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wk, h_wk, D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wv, h_wv, D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wo, h_wo, D * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
    fused_multi_head_attention<<<blocksPerGrid, threadsPerBlock, 2 * T * sizeof(float)>>>(
        d_q, d_k, d_v, d_wq, d_wk, d_wv, d_wo, d_output, T, D
    );

    // Copy results back to host
    hipMemcpy(h_output, d_output, num_elements * sizeof(float), hipMemcpyDeviceToHost);

    // Clean-up
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_wq);
    hipFree(d_wk);
    hipFree(d_wv);
    hipFree(d_wo);
    hipFree(d_output);

    // Print all matrices for validation
    printf("Matrix q:\n");
    for (int i = 0; i < T; ++i) {
        for (int j = 0; j < D; ++j) {
            printf("%.2f ", h_q[i * D + j]);
        }
        printf("\n");
    }

    printf("Matrix k:\n");
    for (int i = 0; i < T; ++i) {
        for (int j = 0; j < D; ++j) {
            printf("%.2f ", h_k[i * D + j]);
        }
        printf("\n");
    }

    printf("Matrix v:\n");
    for (int i = 0; i < T; ++i) {
        for (int j = 0; j < D; ++j) {
            printf("%.2f ", h_v[i * D + j]);
        }
        printf("\n");
    }

    printf("Output matrix:\n");
    for (int i = 0; i < T; ++i) {
        for (int j = 0; j < D; ++j) {
            printf("%.2f ", h_output[i * D + j]);
        }
        printf("\n");
    }

    return 0;
}
