//the optimized matmul lol. I'll copy everything from here once I have it working for once. 
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 

// Tiling size
#define TILE_SIZE 4

__global__ void matMulTiled(float *d_A, float *d_B, float *d_C, int M, int N, int P) {
    __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    // Loop over tiles, N+TILE_SIZE usually has a -1 to account for integer division if it doesn't perfectly line up but we are using an idealized example here
    for (int i = 0; i < (N + TILE_SIZE ) / TILE_SIZE; ++i) {
        // Load elements from A and B to shared memory
        if (row < M && i * TILE_SIZE + threadIdx.x < N)
            ds_A[threadIdx.y][threadIdx.x] = d_A[row * N + i * TILE_SIZE + threadIdx.x];
        else
            ds_A[threadIdx.y][threadIdx.x] = 0.0;

        if (col < P && i * TILE_SIZE + threadIdx.y < N)
            ds_B[threadIdx.y][threadIdx.x] = d_B[(i * TILE_SIZE + threadIdx.y) * P + col];
        else
            ds_B[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Compute inner product for this tile
        #pragma unroll
        for (int j = 0; j < TILE_SIZE; ++j) {
            sum += ds_A[threadIdx.y][j] * ds_B[j][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // Write back result
    if (row < M && col < P)
        d_C[row * P + col] = sum;
}

int main() {
    // Dimension definitions and data setup
    int M = 16, N = 16, P = 16;
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    h_A = (float*)malloc(M * N * sizeof(float));
    h_B = (float*)malloc(N * P * sizeof(float));
    h_C = (float*)malloc(M * P * sizeof(float));

    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_B, N * P * sizeof(float));
    hipMalloc((void**)&d_C, M * P * sizeof(float));

    // Initialize matrices
    // ... (Same as your original code)

    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * P * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((P + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    matMulTiled<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, P);
    hipMemcpy(h_C, d_C, M * P * sizeof(float), hipMemcpyDeviceToHost);

    // print statements (which should also include the print times for the other CUDA stuff)
    printf("Matrix A:\n--------\n");
    for(int i = 0; i < M; ++i) {
            for(int j = 0; j < N; ++j) {
                    printf("%f ", h_A[i * N + j]);
            }
            printf("\n");
    }
      
    printf("--------\n");
    printf("Matrix B:\n--------\n");
    for(int i = 0; i < N; ++i) {
             for(int j = 0; j < P; ++j) {
                    printf("%f ", h_B[i * P + j]);
               }
            printf("\n");
    }
        
    printf("--------\n");
    printf("Matrix C:\n--------\n");
    for(int i = 0; i < M; ++i) {
            for(int j = 0; j < P; ++j) {
                    printf("%f ", h_C[i * P + j]);
            }
            printf("\n");
    }
    printf("--------\n");

    hipFree(d_A); hipFree(d_B); hipFree(d_C); 
    hipFree(h_A); hipFree(h_B); hipFree(h_C);
    return 0;
}
