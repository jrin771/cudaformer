#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 

#define TILE_SIZE 4

__global__ void optimizedMatMul(float *d_A, float *d_B, float *d_C, int M, int N, int P) {
    __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    // Loop over tiles, N+TILE_SIZE usually has a -1 to account for integer division if it doesn't perfectly line up but we are using an idealized example here
    for (int i = 0; i < (N + TILE_SIZE ) / TILE_SIZE; ++i) {
        // Load elements from A and B to shared memory
        if (row < M && i * TILE_SIZE + threadIdx.x < N)
            ds_A[threadIdx.y][threadIdx.x] = d_A[row * N + i * TILE_SIZE + threadIdx.x];
        else
            ds_A[threadIdx.y][threadIdx.x] = 0.0;

        if (col < P && i * TILE_SIZE + threadIdx.y < N)
            ds_B[threadIdx.y][threadIdx.x] = d_B[(i * TILE_SIZE + threadIdx.y) * P + col];
        else
            ds_B[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Compute inner product for this tile. 
        #pragma unroll
        for (int j = 0; j < TILE_SIZE; ++j) {
            sum += ds_A[threadIdx.y][j] * ds_B[j][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    // Write back result
    if (row < M && col < P)
        d_C[row * P + col] = sum;
}
void randomInitialize(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = (float) rand() / RAND_MAX;
    }
}
int main() {
    // Dimension definitions and data setup
    int M = 16, N = 16, P = 16;
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;   
    hipEvent_t start, stop; 
    float elapsedTime; 

    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_A = (float*)malloc(M * N * sizeof(float));
    h_B = (float*)malloc(N * P * sizeof(float));
    h_C = (float*)malloc(M * P * sizeof(float));

    hipEventRecord(start, 0);

    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_B, N * P * sizeof(float));
    hipMalloc((void**)&d_C, M * P * sizeof(float)); 

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to allocate memory on device: %f ms\n", elapsedTime);

    randomInitialize(h_A, M * N); 
    randomInitialize(h_B, N * P); 

    hipEventRecord(start, 0);

    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * P * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to copy data to device: %f ms\n", elapsedTime);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((P + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    hipEventRecord(start, 0);

    optimizedMatMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, P);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to execute kernel: %f ms\n", elapsedTime);

    hipEventRecord(start, 0);

    hipMemcpy(h_C, d_C, M * P * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to copy data from device: %f ms\n", elapsedTime);

    // print statements 
    printf("--------\n");
    printf("Matrix A:\n--------\n");
    for(int i = 0; i < M; ++i) {
            for(int j = 0; j < N; ++j) {
                    printf("%f ", h_A[i * N + j]);
            }
            printf("\n");
    }
      
    printf("--------\n");
    printf("Matrix B:\n--------\n");
    for(int i = 0; i < N; ++i) {
             for(int j = 0; j < P; ++j) {
                    printf("%f ", h_B[i * P + j]);
               }
            printf("\n");
    }
        
    printf("--------\n");
    printf("Matrix C:\n--------\n");
    for(int i = 0; i < M; ++i) {
            for(int j = 0; j < P; ++j) {
                    printf("%f ", h_C[i * P + j]);
            }
            printf("\n");
    }
    printf("--------\n");

    hipFree(d_A); hipFree(d_B); hipFree(d_C); 
    hipFree(h_A); hipFree(h_B); hipFree(h_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
